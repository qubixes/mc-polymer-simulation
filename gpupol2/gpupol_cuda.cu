#include "gpupol_cuda.h"
#include "gpupol.h"
#include "gpupol_cuda_def.h"
#include "ringpol_cu.h"

int GPULibInit(SimProperties* sp, GPUDeviceState* devStates, GPULibContext* cudaContext){
	printf("Initializing GPU\n");
	gpuErrchk( hipGetDeviceCount(&(sp->nDevices)) );
	
	double* perf = (double*)malloc(sizeof(double)*sp->nDevices);
	hipDeviceProp_t prop;
	double maxPerf=0;
	for(int iDev=0; iDev<sp->nDevices; iDev++){
		gpuErrchk( hipGetDeviceProperties(&prop, iDev) );
		perf[iDev] = prop.clockRate*prop.multiProcessorCount;
		if(perf[iDev]>maxPerf) 
			maxPerf=perf[iDev];
	}
	
	int nDev=0;
	for(int iDev=0; iDev<sp->nDevices; iDev++){
		if(perf[iDev]/maxPerf > 0.5)
			cudaContext->devIds[nDev++]=iDev;
	}
	sp->nDevices = nDev;
	free(perf);
	printf("Finished initializing GPU\n");
	return 0;
}

int GPULibLoadBuffers(SimProperties* sp, SimState* ss, GPUDeviceState * devStates, GPULibContext* cudaContext){
	uint globalWs;
	GPUDeviceState* curDev;
	printf("Loading GPU buffers\n");
	globalWs = sp->nwg*sp->ws;
	for(int iDev=0; iDev<sp->nDevices; iDev++){
		curDev = devStates + iDev;
		curDev->latBuf = (char**) malloc(sizeof(char*)*2);
		gpuErrchk( hipSetDevice(cudaContext->devIds[iDev]) );
		
		gpuErrchk( hipMalloc(&(curDev->seedBuf),  sizeof(uint)*globalWs*2*sp->R) );
		gpuErrchk( hipMalloc(&(curDev->latBuf[0]),   sizeof(char)*sp->latSize) );
		gpuErrchk( hipMalloc(&(curDev->latBuf[1]),   sizeof(char)*sp->latSize) );
		gpuErrchk( hipMalloc(&(curDev->transBuf), sizeof(uint)*4) );
		
		gpuErrchk( hipMemcpy(curDev->seedBuf,  ss[iDev].seeds,      sizeof(uint)*globalWs*2*sp->R, hipMemcpyHostToDevice) );
		gpuErrchk( hipMemcpy(curDev->latBuf[0],   ss[iDev].lattice, sizeof(char)*sp->latSize,   hipMemcpyHostToDevice) );
		gpuErrchk( hipMemcpy(curDev->transBuf, sp->trans,           sizeof(uint)*4,                hipMemcpyHostToDevice) );
		curDev->curBuf=0;
	}
	
	for(int iDev=0; iDev<sp->nDevices; iDev++){
		curDev = devStates+iDev;
		hipSetDevice(cudaContext->devIds[iDev]);
		curDev->dt = WLT*(double)RNG_FAC*Rng4(&ss[iDev].rngState);
		curDev->du = WLU*(double)RNG_FAC*Rng4(&ss[iDev].rngState);
		curDev->dv = WLV*(double)RNG_FAC*Rng4(&ss[iDev].rngState);
		
		curDev->cumDt = curDev->dt;
		curDev->cumDu = curDev->du;
		curDev->cumDv = curDev->dv;
		int tuvOff = curDev->dt + curDev->du*WLT+curDev->dv*WLT*WLU;
		polmove <<< sp->nwg, sp->ws, 0 >>> (0, curDev->seedBuf, curDev->latBuf[curDev->curBuf], curDev->latBuf[curDev->curBuf^0x1], curDev->transBuf, 0, tuvOff, sp->nwt, sp->nwu, sp->nwv);
		curDev->curBuf ^= 0x1;
	}
	
	printf("Finished loading buffers\n");
	return 0;
}

int GPULibRelease(SimProperties* sp, GPUDeviceState* devStates, GPULibContext* cudaContext){
	GPUDeviceState* curDev;
	for(int iDev=0; iDev<sp->nDevices; iDev++){
		curDev = devStates+iDev;
		gpuErrchk( hipSetDevice(cudaContext->devIds[iDev]) );
		gpuErrchk( hipFree(curDev->seedBuf) );
		gpuErrchk( hipFree(curDev->latBuf[0]) );
		gpuErrchk( hipFree(curDev->latBuf[1]) );
		gpuErrchk( hipFree(curDev->transBuf) );
	}
	return 0;
}

int GPULibRun(SimProperties* sp, SimState* ss, GPUDeviceState* devStates, GPULibContext* cudaContext, int nTime){
	int iDev;
	int* tuvOff, *prevTuvOff;
	uint NWT, NWU, NWV;
	GPUDeviceState* curDev;
	
	tuvOff = (int*) malloc(sizeof(int)*sp->nDevices);
	prevTuvOff = (int*) malloc(sizeof(int)*sp->nDevices);
	NWT=sp->nwt; NWU=sp->nwu; NWV=sp->nwv;
// 	printf("Starting to run GPU application with %i GPUs\n", sp->nDevices);
	
// 	GetAllRingPolymers();
	for(iDev=0; iDev<sp->nDevices; iDev++){
// 		UpdatePolymerWithLabels(ss+iDev);
		tuvOff[iDev] = devStates[iDev].dt+devStates[iDev].du*WLT+devStates[iDev].dv*WLT*WLU;
	}
// 	nTime=1;
	
// 	for(int i=0; i<sp->latSize; i++){
// 		if(i%sp->LT==0) printf("(%i,%i): ", (i/sp->LT)%sp->LU, i/(sp->LT*sp->LU));
// 		printf("%hhx ", ss[0].lattice[i]);
// 		if(i%sp->LT==sp->LT-1) printf("\n");
// 	}
// 	printf("\n\n\n\n\n");


	for(int i=0; i<nTime; i++){
		for(iDev=0; iDev<sp->nDevices; iDev++){
			curDev = devStates+iDev;
			hipSetDevice(cudaContext->devIds[iDev]);
			
			curDev->dt = WLT*(double)RNG_FAC*Rng4(&ss[iDev].rngState);
			curDev->du = WLU*(double)RNG_FAC*Rng4(&ss[iDev].rngState);
			curDev->dv = WLV*(double)RNG_FAC*Rng4(&ss[iDev].rngState);
			
			curDev->cumDt += curDev->dt;
			curDev->cumDu += curDev->du;
			curDev->cumDv += curDev->dv;
			
			prevTuvOff[iDev] = tuvOff[iDev];
			tuvOff[iDev] = curDev->dt + curDev->du*LCELL*WST + curDev->dv*LCELL*WSU*LCELL*WST;
			
			polmove <<< sp->nwg, sp->ws, 0 >>> (sp->nSteps, curDev->seedBuf, curDev->latBuf[curDev->curBuf], curDev->latBuf[curDev->curBuf^0x1], curDev->transBuf, prevTuvOff[iDev], tuvOff[iDev], NWT, NWU, NWV);
		}
		curDev->curBuf ^= 0x1;
	}
	
	gpuErrchk( hipPeekAtLastError() );
	
	
	for(iDev=0; iDev<sp->nDevices; iDev++){
		curDev = devStates+iDev;
		gpuErrchk( hipSetDevice(cudaContext->devIds[iDev]) );
// 		printf("latSize= %i\n", sp->latSize);
		gpuErrchk( hipMemcpy(ss[iDev].gpuLattice, curDev->latBuf[curDev->curBuf], sizeof(char)*sp->latSize, hipMemcpyDeviceToHost) );
	}
	
	for(iDev=0; iDev<sp->nDevices; iDev++){
		gpuErrchk( hipSetDevice(cudaContext->devIds[iDev]) );
		gpuErrchk( hipDeviceSynchronize() );
	}
	
	for(iDev=0; iDev<sp->nDevices; iDev++){
		CopyGPUToCPULattice(ss[iDev].gpuLattice, ss[iDev].lattice, curDev->cumDt, curDev->cumDu, curDev->cumDv, curDev->dt, curDev->du, curDev->dv, sp);
	}
// 	for(int i=0; i<sp->latSize; i++){
// 		if(i%sp->LT==0) printf("(%i,%i): ", (i/sp->LT)%sp->LU, i/(sp->LT*sp->LU));
// 		printf("%hhx ", ss->gpuLattice[i]);
// 		if(i%sp->LT==sp->LT-1) printf("\n");
// 	}
// 	printf("\n");
// exit(0);

	GetAllRingPolymers();
	for(iDev=0; iDev<sp->nDevices; iDev++)
		UpdatePolymerWithLabels(ss+iDev);
	
	return 0;
}
